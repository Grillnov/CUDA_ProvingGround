#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <string>
#include <sstream>

//#define NDEBUG

#include <CustomUtils\BenchmarkTimerForWindows.h>
#include <CustomUtils\Cusassertion.h>
#include <CustomUtilsCUDA\displayDeviceInfo.h>

const static float eps = 1e-6;
const static size_t blocSize = 8;
const static size_t size = 1024;

__global__ void matMultiply1D(float* matA, float* matB, float* Dest, int dimensions)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i < dimensions)
	{
		float vectA[2048];
		for (unsigned k = 0; k != dimensions; ++k)
		{
			vectA[k] = matB[i*dimensions + k];
		}
		for (unsigned j = 0; j != dimensions; ++j)
		{
			float res = 0.0f;
			for (unsigned k = 0; k != dimensions; ++k)
			{
				res += vectA[k] * matB[k*dimensions + j];
			}
			Dest[i*dimensions + j] = res;
		}
	}
}

__global__ void matMultiply2D(float* matA, float* matB, float* Dest, int dimensions)
{
	int ix = threadIdx.x + blockIdx.x*blockDim.x;
	int iy = threadIdx.y + blockIdx.y*blockDim.y;

	if (ix < dimensions&&iy < dimensions)
	{
		float res = 0.0f;
		for (unsigned k = 0; k != dimensions; ++k)
		{
			res += matA[ix*dimensions + k] * matB[k*dimensions + iy];
		}
		Dest[ix*dimensions + iy] = res;
	}
}

int main(void)
{
	hipError_t err;
	float *A = new float[size*size];
	float *B = new float[size*size];
	float *C = new float[size*size];
	for (unsigned i = 0; i != size; ++i)
	{
		for (unsigned j = 0; j != size; ++j)
		{
			A[i*size + j] = B[i*size + j] = (rand() % 1000) / 1000.0f;
		}
	}

	err = hipSetDevice(0);
	displayDeviceInfo(0);
	float *matA = nullptr;
	float *matB = nullptr;
	float *matC = nullptr;
	err = hipMalloc((void**)&matA, sizeof(float)*size*size);
	err = hipMalloc((void**)&matB, sizeof(float)*size*size);
	err = hipMalloc((void**)&matC, sizeof(float)*size*size);

	err = hipMemcpy(matA, A, sizeof(float)*size*size, hipMemcpyHostToDevice);
	err = hipMemcpy(matB, B, sizeof(float)*size*size, hipMemcpyHostToDevice);

	BenchmarkTimer Benchmarker;
	Benchmarker.startTimer();

	size_t blockCount = size / blocSize;
	if (blockCount - blockCount*blocSize > 0)
	{
		blockCount += 1;
	}
	//matMultiply1D <<<blockCount, blocSize >>>(matA, matB, matC, size);
	dim3 blockSetup(size / blocSize, size / blocSize);
	dim3 blockSizeSetup(blocSize, blocSize);
	matMultiply2D << <blockSetup, blockSizeSetup >> >(matA, matB, matC, size);
	err = hipDeviceSynchronize();

	Benchmarker.endTimer();
	std::cout << "Kernel exec time elapsed: " << Benchmarker.getDeltaTimeInms() << " ms(s)\n";

	err = hipMemcpy(C, matC, sizeof(float)*size*size, hipMemcpyDeviceToHost);
	err = hipGetLastError();
	CusAssertion(err == hipSuccess, "CUDA runtime error caught: " << hipGetErrorString(err));
	std::cout << "Result retrieved" << std::endl;

	Benchmarker.startTimer();
	for (unsigned i = 0; i != size; ++i)
	{
		for (unsigned j = 0; j != size; ++j)
		{
			float res = 0.0f;
			for (unsigned k = 0; k != size; ++k)
			{
				res += A[i*size + k] * B[k*size + j];
			}
			CusAssertion(abs(res - C[i*size + j]) < eps, "res is not equal to C " << i << "," << j);
		}
	}
	Benchmarker.endTimer();
	std::cout << "Host exec time: " << Benchmarker.getDeltaTimeInms() << " ms(s)\n";

	delete[] A;
	delete[] B;
	delete[] C;

	err = hipFree((void**)&matA);
	err = hipFree((void**)&matB);
	err = hipFree((void**)&matC);

	std::cout << "Result correct" << std::endl;

	hipDeviceReset();
}